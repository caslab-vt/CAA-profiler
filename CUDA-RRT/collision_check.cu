#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "collision_check.h"

#include "collision_check.h"



// device global variables
__device__ uint32_t g_max_circles_cell = 100;
__device__ uint32_t g_num_cricles = 1024;
__device__ uint32_t nth_cirlce[100*100];
__device__ uint32_t g_xsize = 100;
__device__ uint32_t g_ysize = 100;
__device__ uint32_t g_bin_size = 100*100*100;

__device__ float g_resolution = 1.0;
__device__ float g_xmin = 0.0;
__device__ float g_xmax = 100.0;
__device__ float g_ymin = 0.0;
__device__ float g_ymax = 100.0;


// kernel declarations
__global__ void binCircles(float3 *c, float3 *bins);

__global__ void kernelSanders1(float *cx, float *cy, float *r, float *q_new, float *q_near, uint32_t *collision_flag);

__global__ void kernelSanders2(float *cx, float *cy, float *r, float *q_new, float *q_near, uint32_t *collision_flag);

__global__ void kernelSanders3(float *cx, float *cy, float *r, float *q_new, float *q_near, uint32_t *collision_flag);



// device functions
__device__ float distance(float cx, float cy, float *qnew);

__device__ float distToCenter(float cx, float cy, float u, float *qnew, float *qnear);

__device__ float composeU(float cx, float cy, float *qnew, float *qnear);

__device__ int world2RowMajor(float x, float y);



// kernels
__global__ void binCircles(float3 *c, float3 *bins)
{
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  // const int numThreads = blockDim.x * gridDim.x;

  const float c_x = c[tid].x;
  const float c_y = c[tid].y;
  const float c_r = c[tid].z;

  __syncthreads();


  int center = world2RowMajor(c_x, c_y);
  int top  = world2RowMajor(c_x, c_y + c_r);
  int left  = world2RowMajor(c_x - c_r, c_y);
  int bottom  = world2RowMajor(c_x, c_y - c_r);
  int right  = world2RowMajor(c_x + c_r, c_y);

  __syncthreads();
  // printf("center: %d\n", center);

    // if (threadIdx.x == 0) printf("center: %d\n", center);
  // printf("[x: %f y: %f r: %f] \n", c_x, c_y, c_r);

  // __syncthreads();
  //
  //
  // for(int i = tid; i < g_bin_size; i += numThreads)
  // {
  //   uint bin_col = atomicInc(&nth_cirlce[center], g_max_circles_cell);
  //   uint bin_index = center * g_max_circles_cell + bin_col;
  //
  //   bins[bin_index] = c[tid];
  //   // printf("bin_index: %u\n", bin_index);
  //
  //   // printf("[x: %f y: %f r: %f] \n", bins[bin_index].x, bins[bin_index].y, bins[bin_index].z);
  //
  // }

  int coords[] = {top, left, bottom, right};
  int uniq[] = {center, -2, -2, -2};
  uint iterator = 1;
  //
  for(int i = 0; i < 4; i++) //iterate through top left right bottom
  {
    for(int j = 0; j < iterator; j++)
    {
      if (coords[i] != uniq[j] && coords[i] >= 0)
      {
        iterator++;
        uniq[iterator] = coords[i];
      }
    }
  }

  __syncthreads();


  for(int i = 0; i < iterator; i++)
  {
  // printf("uniq[iterator]: %u\n", uniq[iterator]);

   uint bin_col = atomicInc(&nth_cirlce[uniq[iterator]], g_max_circles_cell);
   uint bin_index = uniq[iterator] * g_max_circles_cell + bin_col;
   // printf("bin_index: %u\n", bin_index);


   if (tid < g_num_cricles)
   {
     bins[bin_index] = c[tid];
     // printf("tid: %u\n", tid);
   }

   // printf("[x: %f y: %f r: %f] \n", bins[bin_index].x, bins[bin_index].y, bins[bin_index].z);

  }

  __syncthreads();

}



__global__ void kernelSanders1(float *cx, float *cy, float *r, float *q_new, float *q_near, uint32_t *collision_flag)
{


  const int tid = threadIdx.x;

  __shared__ uint32_t flag;
  flag = 0;

  const float c_x = cx[tid];
  const float c_y = cy[tid];
  const float c_r = r[tid];

  const float u = composeU(c_x, c_y, q_new, q_near);
  const float dist_to_ray = distToCenter(c_x, c_y, u, q_new, q_near);
  const float dist_to_q_new = distance(c_x, c_y, q_new);


  // shortest distance to your ray exists in the circle
  if ((dist_to_ray < c_r) && (u < 1) && (u > 0))
  {
    //SET FLAG TO TRUE SHORTEST POINT ON LINE IN CIRLE
    atomicAdd(&flag, 1);
  }


  if(dist_to_q_new < c_r)
  {
    //SET THE FLAG NEW POINT IN CIRCLE
    atomicAdd(&flag, 1);
  }



  __syncthreads();


  // have one thread write result to global memory
  if (tid == 0)
  {
    if (flag > 0)
    {
      *collision_flag = 1;
    }

    else
    {
      *collision_flag = 0;
    }
  }
}



__global__ void kernelSanders2(float *cx, float *cy, float *r, float *q_new, float *q_near, uint32_t *collision_flag)
{


  const int tid = threadIdx.x + blockDim.x * blockIdx.x;

  __shared__ uint32_t flag;
  flag = 0;

  const float c_x = cx[tid];
  const float c_y = cy[tid];
  const float c_r = r[tid];

  const float u = composeU(c_x, c_y, q_new, q_near);
  const float dist_to_ray = distToCenter(c_x, c_y, u, q_new, q_near);
  const float dist_to_q_new = distance(c_x, c_y, q_new);


  // shortest distance to your ray exists in the circle
  if ((dist_to_ray < c_r) && (u < 1) && (u > 0))
  {
    //SET FLAG TO TRUE SHORTEST POINT ON LINE IN CIRLE
    atomicAdd(&flag, 1);
    // *collision_flag = 1;
    // return;
  }


  if(dist_to_q_new < c_r)
  {
    //SET THE FLAG NEW POINT IN CIRCLE
    atomicAdd(&flag, 1);
  }



  __syncthreads();


  // have one thread write result to global memory
  if (tid == 0)
  {
    if (flag > 0)
    {
      *collision_flag = 1;
    }

    else
    {
      *collision_flag = 0;
    }
  }
}



__global__ void kernelSanders3(float3 *bins, float *q_new, float *q_near, uint32_t *collision_flag)
{


  const int tid = threadIdx.x;// + blockDim.x * blockIdx.x;

  __shared__ uint32_t flag;
  flag = 0;

  // const float c_x = c[tid].x;
  // const float c_y = c[tid].y;
  // const float c_r = c[tid].z;

  // use bins to get circle info
  int bin_id = world2RowMajor(q_new[0], q_new[1]); // row into bin

  int bin_index = bin_id * g_max_circles_cell + tid;


  const float c_x = bins[bin_index].x;
  const float c_y = bins[bin_index].y;
  const float c_r = bins[bin_index].z;


  printf("[x: %f y: %f r: %f] \n", bins[bin_index].x, bins[bin_index].y, bins[bin_index].z);


  const float u = composeU(c_x, c_y, q_new, q_near);
  const float dist_to_ray = distToCenter(c_x, c_y, u, q_new, q_near);
  const float dist_to_q_new = distance(c_x, c_y, q_new);


  // shortest distance to your ray exists in the circle
  if ((dist_to_ray < c_r) && (u < 1) && (u > 0))
  {
    //SET FLAG TO TRUE SHORTEST POINT ON LINE IN CIRLE
    atomicAdd(&flag, 1);
    // *collision_flag = 1;
    // return;
  }


  if(dist_to_q_new < c_r)
  {
    //SET THE FLAG NEW POINT IN CIRCLE
    atomicAdd(&flag, 1);
  }



  __syncthreads();


  // have one thread write result to global memory
  if (tid == 0)
  {
    if (flag > 0)
    {
      *collision_flag = 1;
    }

    else
    {
      *collision_flag = 0;
    }
  }
}


__device__ float distance(float cx, float cy, float *qnew)
{
  float dx = cx - qnew[0];
  float dy = cy - qnew[1];
  return sqrt(dx*dx + dy*dy);
}


__device__ float distToCenter(float cx, float cy, float u, float *qnew, float *qnear)
{
  float x = qnew[0] + u*(qnear[0]-qnew[0]);
  float y = qnew[1] + u*(qnear[1]-qnew[1]);
  float p[2] = {x, y};

  return distance(cx, cy, p);
}


__device__ float composeU(float cx, float cy, float *qnew, float *qnear)
{
  float num = (cx-qnew[0])*(qnear[0]-qnew[0]) + (cy-qnew[1])*(qnear[1]-qnew[1]);
  float denom = (qnear[0]-qnew[0])*(qnear[0]-qnew[0]) + (qnear[1]-qnew[1])*(qnear[1]-qnew[1]);
  return num / denom;
}


__device__ int world2RowMajor(float x, float y)
{
  if (!(x >= g_xmin && x <= g_xmax))
  {
    return -1;
  }

  if (!(y >= g_ymin and y <= g_ymax))
  {
    return -1;
  }

  int i = std::floor((x - g_xmin) / g_resolution);
  int j = std::floor((y - g_ymin) / g_resolution);



  if (i == g_xsize)
  {
    i--;
  }

  if (j == g_ysize)
  {
    j--;
  }

  return i * g_xsize + j;
}




void bin_call(float3 *c, float3 *bins, uint32_t mem_size)
{
  hipMemset(bins, 0.0, mem_size*sizeof(float3));

  dim3 dimGrid(1);
  dim3 dimBlock(1024);

  binCircles<<<dimGrid, dimBlock>>>(c, bins);

  hipDeviceSynchronize();
}



void collision_call_1(float *cx, float *cy, float *r, float *q_new, float *q_near, uint32_t *flag)
{
  // set flag to 0
  hipMemset(flag, 0, sizeof(uint32_t));


  dim3 dimGrid(1);
  dim3 dimBlock(1024);

  kernelSanders1<<<dimGrid, dimBlock>>>(cx, cy, r, q_new, q_near, flag);

  hipDeviceSynchronize();
}



void collision_call_2(float *cx, float *cy, float *r, float *q_new, float *q_near, uint32_t *flag, int num_circles)
{
  // set flag to 0
  hipMemset(flag, 0, sizeof(uint32_t));

  int num_blocks = num_circles / 512;

  dim3 dimGrid(num_blocks);
  dim3 dimBlock(512);

  kernelSanders2<<<dimGrid, dimBlock>>>(cx, cy, r, q_new, q_near, flag);

  hipDeviceSynchronize();
}


void collision_call_3(float3 *bins, float *q_new, float *q_near, uint32_t *flag)
{
  // set flag to 0
  hipMemset(flag, 0, sizeof(uint32_t));


  dim3 dimGrid(1);
  dim3 dimBlock(1024);

  kernelSanders3<<<dimGrid, dimBlock>>>(bins, q_new, q_near, flag);

  hipDeviceSynchronize();
}



void copyToDeviceMemory(void* d, void* h, size_t size)
{
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
}


void copyToHostMemory(void* h, void* d, size_t size)
{
	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
}


void* allocateDeviceMemory(size_t size)
{
  void *ptr;
  hipMalloc(&ptr, size);
  return ptr;
}


void freeDeviceMemory(void* d)
{
	hipFree(d);
}
